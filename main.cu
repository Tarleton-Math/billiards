#include "hip/hip_runtime.h"
#include <time.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <random>
#include "/usr/local/cuda/samples/common/inc/hip/hip_vector_types.h"

/***************************************************************
**      STRUCTURE

The main function is called n_body.  Program flow: main -> n_body

**	TAGS

Particles are indexed 0, 1, 2, ..., N-1
Walls are indexed -1, -2, -3, ...
Particles carry tags that indicate which object(s) were involved in its previous collision event.

At each step, for each particle, we compute time to collision with every particle and wall.
However, we do not want to include times for the objects it hit in the previous step.
These times should be exactly zero, but they could have floating point error and appear to be very small
positive numbers.  This will defeat our logic.  Tags are the fix.

max_complex is the largest number of walls a particle may hit simulateously (see HANDLING COMPLEX COLLISIONS)
So each particle carries a list of max_complex tags.  by default, tags are the particle's own index.
The tags list for particle i is written in slots i*max_complex thru (i+1)*max_complex-1 of tag_CP


**     NOTE FOR FUTURE VERSIONS IF WALLS CURVE

In the current version, a particle carries the same tags until its next collision event.
Thus, we continue not computing time to collision with that object after they have separated.
This is fine if all wall are convex, since a particle can not make consecutive collisions
on the same wall or with the same particle.
But for concave walls, this is not true.  We should rethink this logic.


**    HANDLING COMPLEX COLLISIONS   

The logic of this program is greatly complicated by the possibiliiy of "complex collisions" involving
multiple particles and/or multiple walls.  These are quite rare, but can occasionally lead to 
catastrophic failure if not handled.  So we take great care to handle such events.  Sadly, it adds
substantial complexity.  We describe that here.

We allow only 2 type of events
Type I: 1 particle hitting >=1 wall (no additional particles)
	simple: 1 particle, 1 wall
	complex (corner): 1 particle, >=2 walls
Type II: 2 particles hitting each other (no additional particles, no walls)

We allow multiple simultaneous collisions of the above types at different locations (see below).

However, we do not allow collisions that are more complex.  There are 2 types.
Type III: >=2 particles and >=1 walls
jype IV: >=3 particles
(a collision that meets both decriptions will be handled as type III)
We will take care to detect more complex collisions and avoid them as follows.

Def: complex particle = particle involved in a complex collision

Def: randomize = relocate particle randomly in domain so that it is not in contact with any other particle. 

Type III fix: randomize all complex particles
Type IV fix: randomize all complex particles

***     HANDLING TYPE I COMPLEX (CORNER) COLLISIONS

Type I simple = 1 particle, 1 wall.
Type I complex (corner) = 1 particle, >=2 walls.

For corner collisions, we resolve each wall collision separately and sequentially.
However, this could result in a trajectory pointed through a wall and out of the billiard cell.
This could happen if some of the walls uses a non-specular reflection law, or if the corner angle is acute.
To fix this, we loop through the walls again and do another SPECULAR reflection at each
wall that particle's trajectory is pointed through.
We may need to repeat this loop mulitple times if the corner is very acutre.


*** HANDLING MULTIPLE SIMULTANEOUS, SPATIALLY SEPARATED EVENTS

Program handles multiple simultaneous events that are spatially separated.

************************************************************************************/


#ifdef __HIPCC__
#define CUDA_CALLABLE_MEMBER __host__ __device__
#else
#define CUDA_CALLABLE_MEMBER
#endif 

//wall types for collision 
#define passive 0
#define heated  1
#define sliding 2
#define circles 3
#define no_slip 4
#define end 5

#define MIN(x, y) (x < y) ? x : y
#define MAX(x, y) (x > y) ? x : y

#define timetol 0.01
#define hist_length 1000

std :: random_device generator;
std :: uniform_real_distribution<float> unif_dist(0.0, 1.0);
std :: normal_distribution<float> norm_dist(0.0, 1.0);

dim3 block, grid;

//physical parameters
int DIMENSION = 3;
int N = 10;
float MAX_CUBE_DIM = 4.0;
float MIN_CUBE_DIM =-4.0;
float surface_area;
float vol;
float default_radius = 0.5;
float default_mass = 2.0;

//non-physical parameters
bool no_input_file = true;
int MAX_STEPS = 1000;
int FILE_LINES = 1000;
int track_large_particle = 0;
int ignore_particle_interaction = 0;
bool no_slip_particle = true;
int all_particles_diffused = 0;
int diff_number = 2;

//physical constants
float BOLTZ_CONST = 1.0; //1.38064852e-23;
int max_complex = 4;  // number of walls that a particle can hit at a time

//file names for IO
char *in_fname = NULL;
char dir_name[256] = "\0./";

//dynamical variables
float3 *p_CPU = NULL, *p_GPU = NULL; // position
float3 *v_CPU = NULL, *v_GPU = NULL; // velocity
float *angular_momentum = NULL;
float *no_s_gamma = NULL;
float *mass_CPU = NULL, *mass_GPU = NULL; // mass
float *radius_CPU = NULL, *radius_GPU = NULL; // radius
float *p_temp_CPU = NULL; // kinetic energy
float *dt_CPU = NULL, *dt_GPU = NULL; // time particle will take to collide
float t_tot = 0.0;
float default_p_temp = 1.0; // default kinetic energy
float3 collision_normal;

//walls
int num_walls = 0;
float max_temp = 90.0;
float min_temp = 45.0;

//memory management so particle collisions happen in order
int *tag_CPU = NULL, *tag_GPU = NULL;
int *what_w_CPU = NULL, *what_w_GPU = NULL;
int *what_p_CPU = NULL, *what_p_GPU = NULL;
int *how_many_p_CPU = NULL, *how_many_p_GPU = NULL;
int *how_many_w_CPU = NULL, *how_many_w_GPU = NULL;
int complex_colliders = 0;
int *complex_event_particle = NULL;
int *complex_event_log = NULL;
float dt_step = 0.0;


class Wall {
	private: 
		int D = DIMENSION; // for collision methods on GPU
		float max_cube_dim = MAX_CUBE_DIM;
	public: 
		int type; // heated, passive, sliding, or circles
		float2 endpoints[2]; // if a 2D line segment
		float3 position; // center if a circle type wall or position if plane in 3d
		float  radius; // radius if a circle type wall
		float3 normal; // normal vector (assume not a circle at the moment)
		float3 tangent1;
		float3 tangent2;
		float wall_temp[2]; // temperature 
		float alpha; // probability a wall is thermally active
	Wall()
	{
		alpha = 1.0;
		D = DIMENSION;
	}

	// returns 0 or 1 whether a particle (p) collides with the wall (0=no collision)
	// and stores the time until collision in float &time parameter. 
	CUDA_CALLABLE_MEMBER int time_to_collision(float3 p, float3 v, float r, float * time)
	{
		int collides = 0;
		if(type != circles) 
		{
			if(D > 2)
			{
				float tt, max_cube_minus_radius;
				max_cube_minus_radius = max_cube_dim - r;
				tt = -2;
				if( (normal.x > 0.5) && (v.x * v.x > 0.0) )
				{
					tt = ( (-max_cube_minus_radius) - p.x ) / v.x;
				}
				else if( (normal.y > 0.5) && (v.y * v.y > 0.0) )
				{
					tt = ( (-max_cube_minus_radius) - p.y ) / v.y;
				}
				else if( (normal.z > 0.5) && v.z * v.z > 0.0)  
				{
					tt = ( (-max_cube_minus_radius) - p.z ) / v.z;
				}
				else if( (normal.x < -0.5) && (v.x * v.x > 0.0) )
				{
					tt = ( max_cube_minus_radius - p.x ) / v.x;
				}
				else if( (normal.y < -0.5) && (v.y * v.y > 0.0) )
				{
					tt = ( max_cube_minus_radius - p.y ) / v.y;
				}
				else if( (normal.z < -0.5) && (v.z * v.z > 0.0) )
				{
					tt = ( max_cube_minus_radius - p.z ) / v.z;
				}
				if( tt >= 0.0)
				{
				  	(*time) = tt;
					collides = 1;
				}
			}
			else
			{
				float B1,B2,A11,A12,A21,A22,D,t1,t2;
				float x1,y1,x2,y2;

				x1 = endpoints[0].x + (r*normal.x);
				y1 = endpoints[0].y + (r*normal.y);
				x2 = endpoints[1].x + (r*normal.x);
				y2 = endpoints[1].y + (r*normal.y);

				B1 = p.x - x1;
				B2 = p.y - y1;

				A11 = (x2 - x1);
				A12 = -v.x;
				A21 = (y2 - y1);
				A22 = -v.y;

				D = A11 * A22 - (A21 * A12);

				if( abs(D) > 0 )
				{
					t1 = (A22*B1 - A12*B2) / D;
					t2 = (A11*B2 - A21*B1) / D;
					// if they cross at a point in the range of the line segment
					if( (t1 >= 0.0) && (t1 <= 1.0) && (t2 >= 0.0))
					{
						collides = 1;
						(*time) = t2;
					}
				}
			}
		}
		else
		{
			// check circle - vector collision (see GPU code)
		}
		return collides;

	}

	// resolves outgoing particle velocity after collision
	float3 resolve_collision(int p, float3 v)
	{
		float3 v_out;
		if(type == heated)
		{
			float sigma, u, sn, st1, st2, T, m;

			if(DIMENSION > 2)
			{
				T = wall_temp[0];
				m = mass_CPU[p];
				sigma = sqrt(BOLTZ_CONST * T / m);

				u = unif_dist(generator);
				sn = sigma * sqrt(fabs(2.0 * log(1.0 - u)));
				st1 = norm_dist(generator)*sigma;
				st2 = norm_dist(generator)*sigma;
				v_out = sn*normal + st1*tangent1 + st2*tangent2;
			}
			else
			{
				T = wall_temp[0];
				m = mass_CPU[p];
				sigma = sqrt(BOLTZ_CONST*T/m);
				u = unif_dist(generator);
				sn = sigma * sqrt(fabs(2.0*log(1.0-u)));
				st1 = norm_dist(generator)*sigma;

				v_out.x = sn*normal.x + st1 * tangent1.x;
				v_out.y = sn*normal.y + st1 * tangent1.y;
				v_out.z = 0.0;
			}
		}
		else if(type == passive)
		{
			v_out = v - (2.0 * dot(normal, v) * normal);
		}
		else if(type == circles)
		{

		}
		else if(type == no_slip)
		{
			float vn,vt,vn0,vt0,a, g;

			g = no_s_gamma[p]; 
			a = angular_momentum[p]; 

			vn = dot(v_CPU[p], normal);
			vt = dot(v_CPU[p], tangent1);

			angular_momentum[p] 	= -((1.0-g*g)/(1.0+g*g))* a + 
						  -(2.0*g / (1.0+g*g)) 	* vt;

			vt0 = (-2.0*g/(1.0+g*g))  * a + 
			      ((1.0-g*g)/(1.0+g*g)) * vt;

			vn0 = -1.0 * vn;

			v_out = vt0 * tangent1 + vn0 * normal;
		}
		return v_out;
	}
};
Wall * walls_CPU = NULL;
Wall * walls_GPU = NULL;

//Tracks thermodynamical quantities.  We want to maintain a record of the thermodynamical state of system
//over the past "window" of events.  This class helps efficiently compute mean and sd for the current event.
class Thermo_Record {
	private:
		float *X;  // array of most recent values of thermo
		float *X2;  // square of X
		float SX;  // sum of values in X
		float SX2;  // sum of values in SX
		int idx;  // points to the oldest array position. We will overwrite this at the next event.
		int window_size;  // length of arrays X, X2
    

	public:
		float window_mean;  // SX / length(SX)
		float window_sd;  // SX2 / length(SX2) - mean^2
		float latest_value;  //last thing put in 


		Thermo_Record(int n)
		{
			window_size = n;
			SX = SX2 = 0.0;
			idx = 0;
			window_mean = window_sd = 0.0;
			latest_value = 0.0;
			X = (float*)calloc(window_size, sizeof(float));
			X2 = (float*)calloc(window_size, sizeof(float));
		}

		~Thermo_Record()
		{
			SX = SX2 = window_mean = window_sd = 0.0;
			if(X != NULL) { free(X); }
			if(X != NULL) { free(X2); }
		}

		void update(float value)
		{
			SX -= X[idx];
			X[idx] = value;
			SX += value;

			window_mean = SX / window_size;

			SX2 -= X2[idx];
			X2[idx] = value * value;
			SX2 += X2[idx];

			window_sd = SX2 / window_size - window_mean * window_mean;
			idx = ( (idx + 1 ) >= window_size) ? 0 : idx + 1;

			latest_value = value;
		}
};


float impulse_sum = 0.0;
float heat_sum = 0.0;
float entropy_sum = 0.0;
int thermo_rec_length = 1000;
Thermo_Record pressure = Thermo_Record(thermo_rec_length);
Thermo_Record gas_temperature = Thermo_Record(thermo_rec_length);

void compute_thermodynamics(float3 v_in, float3 v_out, float3 normal, float wall_temp, float mass, float t_tot)
{
	float impulse, P;
	impulse = mass * ( dot(v_out - v_in, normal));
	impulse_sum += impulse;
	P = (impulse_sum / t_tot) / surface_area;
	pressure.update(P);

	float gas_temp = 0.0;
	for(int i = 0; i < N; i++)
	{
		gas_temp += mass_CPU[i] * dot(v_CPU[i], v_CPU[i]) 
				/ (1.0 * DIMENSION * BOLTZ_CONST);
	}
	gas_temp /= (1.0 * N);
	gas_temperature.update(gas_temp);

	float q_in = mass * dot(v_in, v_in);
	float q_out = mass * dot(v_out, v_out);
	float dq = q_out - q_in;

	heat_sum += dq;
	entropy_sum +=dq/wall_temp;
}


void make_orthonormal_frame(float3 * n, float3 * t1, float3 * t2)
{

	if( ( (*n).x * (*n).x > 0) || ((*n).y * (*n).y > 0) )
	{
		(*t1).x = (*n).y;
		(*t1).y =-(*n).x;
		(*t1).z = 0;
	}
	else if((*n).z * (*n).z > 0)
	{
		(*t1).x =-(*n).z;
		(*t1).y = 0;
		(*t1).z = (*n).x;
	}
	else
	{
		printf("Failed to initialize normal and tangent vectors");
		exit(1);
	}
	(*t2) = cross((*n), (*t1));
	(*n) = normalize(*n);
	(*t1) = normalize(*t1);
	(*t2) = normalize(*t2);
}		



void allocate_wall_memory()
{
	walls_CPU = (Wall*)malloc(num_walls * sizeof(Wall));
	for(int i = 0; i < num_walls; i++) walls_CPU[i] = Wall();
}

void setup_walls()
{
	int i;

	surface_area = 6*(2*MAX_CUBE_DIM)*(2*MAX_CUBE_DIM);
	vol = (2*MAX_CUBE_DIM)*(2*MAX_CUBE_DIM)*(2*MAX_CUBE_DIM);

	for(i = 0; i < 6; i++)
	{
		max_temp = MAX(max_temp, walls_CPU[i].wall_temp[0]);
		min_temp = MIN(min_temp, walls_CPU[i].wall_temp[0]);
	}
	default_p_temp = (max_temp + min_temp) / 2.0;

}

// returns true if particle is inside domain, false if outside
bool inside_domain(float3 p, float r)
{
	int num_total;

	if(DIMENSION > 2)
	{
		r*=1.5;
		if( 	(p.x*p.x > ((MAX_CUBE_DIM-r)*(MAX_CUBE_DIM-r))) || 
			(p.y*p.y > ((MAX_CUBE_DIM-r)*(MAX_CUBE_DIM-r))) || 
			(p.z*p.z > ((MAX_CUBE_DIM-r)*(MAX_CUBE_DIM-r))) )
		{
			num_total = 2;
		}
		else
		{
			num_total = 1;
		}
	}
	else
	{
		num_total = 0;
		float t, d = (DIMENSION > 2) ? 1.0 : 0.0;
		float3 v = make_float3(2, 2, 2*d);

		for(int w = 0; w < num_walls; w++)
		{
			num_total += walls_CPU[w].time_to_collision(p, v, r, &t);
		}
	}
	return (num_total%2);
}

// find position chosen randomly in box for particle such that 
// (1) the new position is within the domain
// (2) particle at new position is not in contact / overlapping with any other particles. 
void randomize_position(int p, float MIN_EXTENT, float MAX_EXTENT)
{
	float px, py, pz, dd;
	float3 new_pos;
	bool needs_new_position = true;

	while(needs_new_position)
	{
		needs_new_position = false;

		// create new x, y, z coordinate for particle p
		px = MIN_EXTENT + (MAX_EXTENT-MIN_EXTENT) * unif_dist(generator);
		py = MIN_EXTENT + (MAX_EXTENT-MIN_EXTENT) * unif_dist(generator);
		pz = 0.0;
		if(DIMENSION > 2) 
		{
			pz = MIN_EXTENT + (MAX_EXTENT - MIN_EXTENT) * unif_dist(generator);
		}
		new_pos = make_float3(px, py, pz);

		// check if it is inside domain AND doesn't overlap with any other particles. 
		if( inside_domain(new_pos, radius_CPU[p]) )
		{
			for(int i = 0; i < N; i++)
			{
				dd = dot(p_CPU[i] - new_pos, p_CPU[i] - new_pos);
				if(dd < (radius_CPU[i] + radius_CPU[p]) * (radius_CPU[i] + radius_CPU[p]) )
				{
					needs_new_position = true;
					i = N;
				}
			}
		}
		else
		{
			needs_new_position = true;
		}
	}
	p_CPU[p] = new_pos;
	printf("Looking for positions for particle %d - %lf %lf %lf\n", p,p_CPU[p].x,p_CPU[p].y,p_CPU[p].z);
}


void distribute_particles(float MIN_EXTENT, float MAX_EXTENT, int DIM)
{
	float T, d = (DIM > 2) ? 1.0 : 0.0;
	int i;

	//set initial particle parameters
	for (i = 0; i < N; i++)
	{
		mass_CPU[i] = default_mass;
		radius_CPU[i] = default_radius;
		p_temp_CPU[i] = default_p_temp;
		no_s_gamma[i] = 1.0/sqrt(2.0);
		angular_momentum[i] = 0.0;
	}

	if (track_large_particle)
	{
		radius_CPU[0] = 3.0 * radius_CPU[0];
		mass_CPU[0] = 3.0 * mass_CPU[0];
	}

	for (i = 0; i < N; i++)
	{
		T = sqrt(BOLTZ_CONST * p_temp_CPU[i] / mass_CPU[i]);
		v_CPU[i].x = norm_dist(generator)*T;
		v_CPU[i].y = norm_dist(generator)*T;
		if(DIMENSION > 2) v_CPU[i].z = norm_dist(generator)*T;
		else v_CPU[i].z = 0.0;
	}

	for(i = 0; i < N; i++) p_CPU[i] = 2.0 * MAX_EXTENT * make_float3(1.0, 1.0, d);
	for(i = 0; i < N; i++) randomize_position(i, MIN_EXTENT, MAX_EXTENT);
}

void allocate_CPU_memory()
{
	p_CPU			= (float3*)malloc(		N * sizeof(float3) );
	v_CPU			= (float3*)malloc(		N * sizeof(float3) );
	angular_momentum 	= (float* )malloc(      	N * sizeof(float ) );
	no_s_gamma          	= (float* )malloc(		N * sizeof(float ) );
	radius_CPU		= (float* )malloc(		N * sizeof(float ) );
	mass_CPU		= (float* )malloc(		N * sizeof(float ) );
	dt_CPU			= (float* )malloc(		N * sizeof(float ) );
	p_temp_CPU		= (float* )malloc(		N * sizeof(float ) );
	tag_CPU			= (int*   )malloc(max_complex *	N * sizeof(int   ) );
	how_many_p_CPU 		= (int*   )malloc(		N * sizeof(int   ) );
	how_many_w_CPU 		= (int*   )malloc(		N * sizeof(int   ) );
	what_p_CPU		= (int*   )malloc(max_complex *	N * sizeof(int   ) );
	what_w_CPU		= (int*   )malloc(max_complex *	N * sizeof(int   ) );
	complex_event_particle 	= (int*   )malloc(		N * sizeof(int   ) );
	complex_event_log 	= (int*   )malloc(2 *		N * sizeof(int   ) );

	//tag particles not to hit themselves
	for(int i = 0; i < max_complex * N; i++) tag_CPU[i] = N + 2;
}

void allocate_GPU_memory()
{

	// GPU MEMORY ALLOCATION
	block.x = 1024;
	block.y = 1;
	block.z = 1;

	grid.x = (N - 1) / block.x + 1;
	grid.y = 1;
	grid.z = 1;

	hipMalloc( (void**)&p_GPU,       N *sizeof(float3) );
	hipMalloc( (void**)&v_GPU,       N *sizeof(float3) );
	hipMalloc( (void**)&radius_GPU,  N *sizeof(float ) );
	hipMalloc( (void**)&mass_GPU,    N *sizeof(float ) );

	hipMalloc( (void**)&tag_GPU,	  max_complex * N *sizeof(int  ) );
	hipMalloc( (void**)&dt_GPU,			N *sizeof(float) );
	hipMalloc( (void**)&how_many_p_GPU,		N *sizeof(int  ) );
	hipMalloc( (void**)&how_many_w_GPU,		N *sizeof(int  ) );
	hipMalloc( (void**)&what_p_GPU,  max_complex * N *sizeof(int  ) );
	hipMalloc( (void**)&what_w_GPU,  max_complex * N *sizeof(int  ) );

	hipMalloc(&walls_GPU, num_walls*sizeof(Wall));

	// copy CPU initialization to GPU
	hipMemcpy( p_GPU,      p_CPU,                  N *sizeof(float3), hipMemcpyHostToDevice );
	hipMemcpy( v_GPU,      v_CPU,                  N *sizeof(float3), hipMemcpyHostToDevice );
	hipMemcpy( mass_GPU,   mass_CPU,               N *sizeof(float ), hipMemcpyHostToDevice );
	hipMemcpy( radius_GPU, radius_CPU,             N *sizeof(float ), hipMemcpyHostToDevice );
	hipMemcpy( tag_GPU,    tag_CPU,  max_complex * N *sizeof(int   ), hipMemcpyHostToDevice );

	hipMemcpy(walls_GPU, walls_CPU, num_walls*sizeof(Wall), hipMemcpyHostToDevice);
}


// reads input file of parameters (temperature of walls, particle size, &c)
void read_input_file_and_set_initial_conditions()
{
  	FILE * fp = NULL;
	const int bdim = 132;
	char buff[bdim];
	int i, d, dim, n;
	double f, g, h, f1, g1, h1, t1, t2, v1, v2, v3;
	char s[256];
	double MINCD,MAXCD;
	bool particle_initialization_by_file = false;

	if(no_input_file)
	{
		printf("No input file. Using default values.\n");
		num_walls = 6;
		allocate_wall_memory();

		walls_CPU[0].normal.x = 1.0; walls_CPU[0].normal.y = 0.0; walls_CPU[0].normal.z = 0.0;
		walls_CPU[2].normal.x = 0.0; walls_CPU[2].normal.y = 1.0; walls_CPU[2].normal.z = 0.0;
		walls_CPU[4].normal.x = 0.0; walls_CPU[4].normal.y = 0.0; walls_CPU[4].normal.z = 1.0;
		walls_CPU[1].normal.x =-1.0; walls_CPU[1].normal.y = 0.0; walls_CPU[1].normal.z = 0.0;
		walls_CPU[3].normal.x = 0.0; walls_CPU[3].normal.y =-1.0; walls_CPU[3].normal.z = 0.0;
		walls_CPU[5].normal.x = 0.0; walls_CPU[5].normal.y = 0.0; walls_CPU[5].normal.z =-1.0;

		for(i = 0; i < 6; i++)
		{
			walls_CPU[i].type=passive;
			make_orthonormal_frame(&(walls_CPU[i].normal), &(walls_CPU[i].tangent1), &(walls_CPU[i].tangent2));
			walls_CPU[i].position = -MAX_CUBE_DIM * (walls_CPU[i].normal);
		}
		allocate_CPU_memory();
		distribute_particles(MIN_CUBE_DIM, MAX_CUBE_DIM, DIMENSION);
		allocate_GPU_memory();
	}
	else
	{
		fp = fopen(in_fname,"r");

		fgets(buff,bdim,fp);
		fgets(buff,bdim,fp);
		//sscanf(buff, "%d", &dim); DIMENSION = dim;
		sscanf(buff, "%d", &DIMENSION);

		fgets(buff, bdim, fp);
		fgets(buff, bdim, fp);
		//sscanf(buff, "%d", &n);
		//N = n;
		sscanf(buff, "%d", &N);
		allocate_CPU_memory();

		fgets(buff, bdim, fp);
		fgets(buff, bdim, fp);
		//sscanf(buff, "%lf", &f); default_radius = f;
		sscanf(buff, "%f", &default_radius);
		if(default_radius > 0){ }
		else { ignore_particle_interaction = true; }

		fgets(buff,bdim,fp);
		fgets(buff,bdim,fp);
		//sscanf(buff, "%d", &d); MAX_STEPS = d;
		sscanf(buff, "%d", &MAX_STEPS);

		fgets(buff, bdim, fp);
		fgets(buff, bdim, fp);
		sscanf(buff, "%d", &num_walls);
		allocate_wall_memory();

		if(DIMENSION < 3) MINCD = MAXCD = 0.0;

		fgets(buff, bdim, fp);
		for(i = 0; i < num_walls; i++)
		{
			fgets(buff, bdim, fp);
			sscanf(buff, "%d %lf %lf %lf %lf %lf %lf %lf %lf",&d,&t1,&t2,&f,&g,&h,&f1,&g1,&h1);

			walls_CPU[i].wall_temp[0] = t1;
			walls_CPU[i].wall_temp[1] = t2;

			walls_CPU[i].type = d;

			if(d != circles)
			{
				if(DIMENSION < 3)
				{
					walls_CPU[i].endpoints[0].x = f;
					walls_CPU[i].endpoints[0].y = g;
					walls_CPU[i].endpoints[1].x = f1;
					walls_CPU[i].endpoints[1].y = g1;
					// inward pointing normals assumes that the walls are given in clockwise orientation. 
					walls_CPU[i].normal = normalize(make_float3((g1-g),(f-f1),0.0));
					walls_CPU[i].tangent1 = normalize(make_float3((f-f1),(g-g1),0.0));
					walls_CPU[i].tangent2 = make_float3(0.0,0.0,0.0);

					MAXCD = MAX(MAXCD, f);
					MAXCD = MAX(MAXCD, g);
					MAXCD = MAX(MAXCD, f1);
					MAXCD = MAX(MAXCD, g1);
					MINCD = MIN(MINCD, f);
					MINCD = MIN(MINCD, g);
					MINCD = MIN(MINCD, f1);
					MINCD = MIN(MINCD, g1);
				}
				else
				{
					walls_CPU[i].position = make_float3(f1,g1,h1);
					walls_CPU[i].normal = normalize(make_float3(f,g,h));
					make_orthonormal_frame(&(walls_CPU[i].normal), 
								&(walls_CPU[i].tangent1), 
								&(walls_CPU[i].tangent2));

					MINCD = MIN(MINCD, f);
					MINCD = MIN(MINCD, g);
					MINCD = MIN(MINCD, h);
					MINCD = MIN(MINCD, f1);
					MINCD = MIN(MINCD, g1);
					MINCD = MIN(MINCD, h1);

					MAXCD = MAX(MAXCD, f);
					MAXCD = MAX(MAXCD, g);
					MAXCD = MAX(MAXCD, h);
					MAXCD = MAX(MAXCD, f1);
					MAXCD = MAX(MAXCD, g1);
					MAXCD = MAX(MAXCD, h1);
				}
			}
			else
			{
				walls_CPU[i].position = make_float3(f,g,h);
				walls_CPU[i].radius = f1;
			}
		}

		fgets(buff, bdim, fp);
		fgets(buff, bdim, fp);
		sscanf(buff, "%d", &d);
		no_slip_particle = (d > 0) ? true : false;

		fgets(buff, bdim, fp);
		fgets(buff, bdim, fp);
		sscanf(buff, "%d", &particle_initialization_by_file);
		if(particle_initialization_by_file)
		{
			for(i = 0; i < N; i++)
			{
				fgets(buff, bdim, fp);
				sscanf(buff, "%lf %lf %lf %lf %lf %lf %lf %lf %lf %lf", 
				&f1, &t1, &f, &g, &h, &g1, &h1, &v1, &v2, &v3);

				angular_momentum[i] = f1;
				no_s_gamma[i] = t1;
				radius_CPU[i] = g1;
				mass_CPU[i] = h1;
				p_CPU[i] = make_float3(f, g, h);
				v_CPU[i] = make_float3(v1, v2, v3);
			}
		}
		else
		{
			distribute_particles(MINCD, MAXCD, DIMENSION);
		}
		allocate_GPU_memory();

		fgets(buff, bdim, fp);
		fgets(buff, bdim, fp);
		sscanf(buff, "%s", s);
		strcpy(dir_name, s);

		MIN_CUBE_DIM = MINCD;
		MAX_CUBE_DIM = MAXCD;
	}
}


float get_intersection_point(float time, int p)
{
	return p_CPU[p].x + v_CPU[p].x * time;
}


__device__ int particle_particle_collision(float3 * p, float3 * v, float * radius, int i0, int i1, float * t)
{
	float xc, yc, zc, xv, yv, zv;
	float discriminant, dd, a, b, c, dt;
	int collides = 0;

	dd = (radius[i0] + radius[i1]) * (radius[i0] + radius[i1]);

	xc = p[i0].x - p[i1].x;
	yc = p[i0].y - p[i1].y;
	zc = p[i0].z - p[i1].z;

	if( (xc * xc + yc * yc + zc * zc) > dd){

		xv = v[i0].x - v[i1].x;
		yv = v[i0].y - v[i1].y;
		zv = v[i0].z - v[i1].z;

		a = xv * xv + yv * yv + zv * zv;
		b = 2.0 * (xc * xv + yc * yv + zc * zv);
		c = (xc * xc) + (yc * yc) + (zc * zc) - dd;

		discriminant = b * b - (4.0 * a * c);

		if(discriminant >= 0.0)
		{
			if(a * a > 0.0) // solve ax^2 + bx + c = 0
			{
				// choose the smallest positive root
				dt = (-b - sqrt(discriminant)) / (2.0 * a);
				if(dt < 0.0)
				{
					dt = (-b + sqrt(discriminant)) / (2.0 * a);
				}
			}
			else if(b * b > 0) // solve bx + c = 0 
			{
				dt = -c / b;
			}
			else if(c * c > 0)
			{
				dt = 0.0;
			}
			else
			{
				dt = -1.;
			}
			if(dt >= 0.0)
			{
				collides = 1;
				*t = dt;
			}
		}
	}
	return collides;
}



__global__ void find_dts(float3 * p, float3 * v, float * radius, float * mass,  Wall * w, // particle data--position, velocity, radius 
			int * tag, int * how_many_p, int * how_many_w, int * what_p_hit, int * what_w_hit, // memory management--what each particle hits and has hit
			int n, int max_complex, int ignore_particle_interaction, float * min_dt) // macros--number of particles, shape of geometry, &c.
{
	float dt, current_min_dt = 0.0;
	int j, k, ok, collides, first_collision, this_particle;
	float3 this_p, this_v;

	first_collision = 1;
	current_min_dt = 20000000;

	this_particle = blockDim.x * blockIdx.x + threadIdx.x;

	if(this_particle < n)
	{
		this_p = p[this_particle];
		this_v = v[this_particle];

		how_many_p[this_particle] = 0;
		how_many_w[this_particle] = 0;

		first_collision = 1;

		if(ignore_particle_interaction)
		{
			// do nothing-only interested in particle-wall collisions
		}
		else
		{
			// check current particle against all particles for collision
			for(j = 0; j < n; j++)			
			{
				ok = 1;
				if((this_particle == j) || ((tag[max_complex * this_particle] == j) && (tag[max_complex * j] == this_particle)) )ok = 0;
				if(ok > 0)
				{
				  	collides = particle_particle_collision(p, v, radius, this_particle, j, &dt);
					if(collides > 0)
					{
						if(first_collision > 0)
						{	
							current_min_dt = dt;
							what_p_hit[max_complex*this_particle] = j;
							first_collision = 0;
							how_many_p[this_particle] = 1;
						}
						else
						{
							if( dt < current_min_dt )
							{
								current_min_dt = dt;
								what_p_hit[max_complex*this_particle] = j;
								how_many_p[this_particle] = 1;
							}
							else if( dt <= current_min_dt)
							{
								current_min_dt = dt;
								what_p_hit[max_complex*this_particle+how_many_p[this_particle]] = j;
								how_many_p[this_particle]++;
							}
						}
					}
				}
			}
		}
		

		// check current particle against walls for collision
		how_many_w[this_particle] = 0;
		for(j = 0; j < 6; j++)
		{
			ok = 1;
			for(k = 0; k < max_complex; k++) 
			{
				if(tag[max_complex * this_particle + k] == -(j + 1) )
				{
					ok = 0;
				}
			}
			if(ok > 0)
			{
				collides = w[j].time_to_collision(p[this_particle],v[this_particle],radius[this_particle],&dt);
				if( collides > 0 )
				{
					if(first_collision > 0)
					{
						current_min_dt = dt;
						what_w_hit[max_complex * this_particle] = -(j + 1);
						first_collision = 0;
						how_many_w[this_particle] = 1;
						how_many_p[this_particle] = 0;
					}
					else
					{
						if( dt < current_min_dt)
						{
							current_min_dt = dt;
							what_w_hit[max_complex * this_particle] = -(j + 1);
							how_many_w[this_particle] = 1;
							how_many_p[this_particle] = 0;
							
						}
						else if( dt <= current_min_dt)
						{
							what_w_hit[max_complex * this_particle + how_many_w[this_particle]] = -(j + 1);
							how_many_w[this_particle]++;
						}
							
					}
				}
			}
		}
		min_dt[this_particle] = current_min_dt;
	}
}


float3 specular_reflect(float3 v_in, float3 n)
{
	float3 v_out = v_in - (2.0 * dot(n, v_in) * n);
	return(v_out);
}

float chi_sq(float u, float sigma)
{
	return sigma * sqrt(fabs(2.0 * log(1.0 - u)));
}


float3 heated_wall_reflection(float3 v_in, float3 n, float3 t1, float3 t2, float T, float m)
{
	float3 v_out;
	float u, sn, st1, st2;
	float sigma = sqrt(BOLTZ_CONST * T / m);

	u = unif_dist(generator);
	sn = chi_sq(u, sigma);
	st1 = norm_dist(generator)*sigma;
	st2 = norm_dist(generator)*sigma;

	v_out = sn*n + st1*t1 + st2*t2;
	return v_out;
}

void elastic_particle_particle_collision(int i1, int i2)
{
	float3 n, v1_n, v1_t, v2_n, v2_t;
	float s1_n, s2_n, m1, m2, M;

	tag_CPU[max_complex * i1] = i2;
	tag_CPU[max_complex * i2] = i1;

	m1 = mass_CPU[i1];
	m2 = mass_CPU[i2];
	M = m1 + m2;

	// n is the vector from center of i0 to center of i1
	n = normalize(p_CPU[i1] - p_CPU[i2]);

	//--------------------Particle 1-----------------------//
	// get vector component of v_CPU[i0] parallel to n
	s1_n = dot(n, v_CPU[i1]);
	v1_n = s1_n * n;
	// get vector component of v_CPU[i0] perpendicular to n
	v1_t = v_CPU[i1] - v1_n;
	//-----------------------------------------------------//


	//--------------------Particle 2-----------------------//
	// get vector component of v_CPU[i1] parallel to n
	s2_n = dot(n, v_CPU[i2]);
	v2_n = s2_n * n;
	// get vector component of v_CPU[i1] perpendicular to n
	v2_t = v_CPU[i2] - v2_n;
	//-----------------------------------------------------//

	// update velocities
	v_CPU[i1] = v1_t + (((m1-m2)*s1_n + (2*m2 )*s2_n)/M) * n;
	v_CPU[i2] = v2_t + (((2*m1 )*s1_n + (m2-m1)*s2_n)/M) * n;
}

void no_slip_particle_particle_collision(int i1, int i2)
{
	float3 n, t;
	float m,m1,m2,delta,vn1,vt1,vn2,vt2,a1,a2;
	float a11,t11,n11,a22,t22,n22, g1,g2;

	g1 = no_s_gamma[i1]; g2 = no_s_gamma[i2];
	m1 = mass_CPU[i1]; m2 = mass_CPU[i2]; m = m1+m2;

	a1 = angular_momentum[i1]; a2 = angular_momentum[i2];

	delta = 2.0 / ( (1. + 1. / (g1*g1)) / m1 + 
			(1. + 1. / (g2*g2)) / m2);

	n = normalize(p_CPU[i1] - p_CPU[i2]);
	t = make_float3(n.y, n.x, 0.0);

	vn1 = dot(v_CPU[i1], n);
	vt1 = dot(v_CPU[i1], t);
	vn2 = dot(v_CPU[i2], n);
	vt2 = dot(v_CPU[i2], t);

	a11 = 	(1.0 - delta / (m1*g1*g1)) 	* a1 + 
		(delta / (m1*g1)) 		* vt1 + 
		0.0 				* vn1 + 
		(-delta / (m1*g1*g2)) 		* a2 + 
		(-delta / (m1*g1))		* vt2 + 
		0.0 				* vn2;

	t11 = 	(delta / (m1*g1))		* a1 + 
		(1.0 - delta / m1) 		* vt1 + 
		0.0				* vn1 + 
		(delta / (m1*g2))		* a2 + 
		(delta / m1)			* vt2 + 
		0.0 				* vn2;

	n11 = 	0.0 				* a1 + 
		0.0 				* vt1 + 
		(1.0 - 2.0*m2 / m)		* vn1 + 
		0.0 				* a2 + 
		0.0 				* vt2 + 
		(2.0 * m2 / m) 			* vn2;

	a22 = 	(-delta / (m2*g1*g2))	 	* a1 + 
		(delta / (m2*g2)) 		* vt1 + 
		0.0 				* vn1 + 
		(1.0 - delta / (m2*g2*g2)) 	* a2 + 
		(-delta / (m2*g2))		* vt2 + 
		0.0 				* vn2;

	t22 = 	(-delta / (m2*g1))		* a1 + 
		(delta / m2) 			* vt1 + 
		0.0				* vn1 + 
		(-delta / (m2*g2))		* a2 + 
		(1.0 - delta / m2)		* vt2 + 
		0.0 				* vn2;

	n22 = 	0.0 				* a1 + 
		0.0 				* vt1 + 
		(2.0*m1 / m)			* vn1 + 
		0.0 				* a2 + 
		0.0 				* vt2 + 
		(1.0 - 2.0 * m1 / m) 		* vn2;

	angular_momentum[i1] = a11;
	angular_momentum[i2] = a22;

	v_CPU[i1] = t11 * t + n11 * n;
	v_CPU[i2] = t22 * t + n22 * n;

	tag_CPU[max_complex * i1] = i2;
	tag_CPU[max_complex * i2] = i1;
}


void errorCheck(int num, const char * message)
{
	hipError_t error;
	error = hipGetLastError();
	if(error != hipSuccess)
	{
		printf("Cuda Error at time %d: %s = %s\n", num, message, hipGetErrorString(error));
	}
}
void add_recursively_to_complex_event_particles(int p, int ignored_val)
{
	bool not_included_yet = true;
	int num_p = 0, num_w = 0;
	if(p != ignored_val)
	{
		for(int i = 0; i < complex_colliders; i++)
			if(complex_event_particle[i] == p)
				not_included_yet = false;
		if(not_included_yet)
		{
			complex_event_particle[complex_colliders] = p;
			complex_colliders++;
		}
		num_p = how_many_p_CPU[p];
		num_w = how_many_w_CPU[p];
		how_many_p_CPU[p] = how_many_w_CPU[p] = 0;
		for(int i = 0; i < num_p; i++)
			add_recursively_to_complex_event_particles(what_p_CPU[max_complex*p+i], ignored_val);

		complex_event_log[2*complex_colliders  ] = num_p;
		complex_event_log[2*complex_colliders+1] = num_w;
	}
}

bool detect_collision_events()
{
	bool anything_complex_found = false;
	int i, j, k;

	//find global min dt
	dt_step = dt_CPU[0];
	for (i = 1; i < N; i++) if(dt_CPU[i] <= dt_step) dt_step = dt_CPU[i];

	complex_colliders = 0;
	for(i = 0; i < N; i++)
	{
		if(dt_CPU[i] > dt_step) how_many_p_CPU[i] = how_many_w_CPU[i] = 0;

		if(how_many_p_CPU[i] > 1)
		{
			add_recursively_to_complex_event_particles(i, -2);
			anything_complex_found = true;
		}
		else if( (how_many_p_CPU[i] > 0) & (how_many_w_CPU[i] > 0) )
		{
			for(j = 0; j < how_many_p_CPU[i]; j++)
			{
				k = what_p_CPU[max_complex*i+j];
				add_recursively_to_complex_event_particles(k, i);
			}
			anything_complex_found = true;
		}
	}
	return anything_complex_found;
}

void check_no_particles_escape(int time_step, float total_time)
{
	for (int i = 0; i < N; i++)
	{
		if(     ((p_CPU[i].x*p_CPU[i].x) > (MAX_CUBE_DIM * MAX_CUBE_DIM)) || 
			((p_CPU[i].y*p_CPU[i].y) > (MAX_CUBE_DIM * MAX_CUBE_DIM)) || 
			((p_CPU[i].z*p_CPU[i].z) > (MAX_CUBE_DIM * MAX_CUBE_DIM)) ) 
		{
			printf("Error at time step %d\t physical time %f\t particle %d escaped\n", time_step, total_time, i);
			printf("\tit hit (%d %d) -> %d %d %d\n", how_many_p_CPU[i], how_many_w_CPU[i], 
								tag_CPU[max_complex*i+0], 
								what_w_CPU[max_complex*i+0], 
								what_p_CPU[max_complex*i+1]); 
			printf("\tNew pos: (%f %f %f)\n", p_CPU[i].x, p_CPU[i].y, p_CPU[i].z);
			fflush(stdout);
			exit(1);
		}
	}
}

void print_output_file(FILE *fp, int i, int j, int thisstep, float time, float3 collision_normal)
{
	int k;
	if(thisstep < 0)
	{
		if(DIMENSION < 3)
		{
			for(int jj = 0; jj < N; jj++) fprintf(fp, "%lf %lf %lf ", p_CPU[jj].x, p_CPU[jj].y, radius_CPU[jj]);
				fprintf(fp, "%lf\n", time);
			/*
			float ddt = 0.0;
			int num = int(time/timetol);
			num = (num > 0) ? num : 1;
			for(int jj = 0; jj < num; jj++)
			{
				for(k = 0; k < N; k++) 
				{
					fprintf(fp, "%lf %lf %lf ", 
							p_CPU[k].x + (ddt * v_CPU[k].x), 
							p_CPU[k].y + (ddt * v_CPU[k].y), 
							radius_CPU[k]);
				}
				fprintf(fp, "\n");
				ddt+=timetol;
			}
			*/
		}
	}
	else if(thisstep < 1)
	{
		if(DIMENSION > 2)
		{
			fprintf(fp, "#box dimension\n box, %lf\n", MAX_CUBE_DIM);
			fprintf(fp, "#particle radii\n");
			for(k = 0; k < N; k++) fprintf(fp, "r, %d, %lf, %lf\n", k, radius_CPU[k], mass_CPU[k]);
		}
		else
		{
			FILE *bash_script = NULL;
			bash_script = fopen("edge_file","w");
			for(k = 0; k < num_walls; k++) 
			{
				fprintf(bash_script, "%lf %lf\n%lf %lf\n", 
					walls_CPU[k].endpoints[0].x, 
					walls_CPU[k].endpoints[0].y, 
					walls_CPU[k].endpoints[1].x, 
					walls_CPU[k].endpoints[1].y 
					);
			}
			fclose(bash_script);

			bash_script = fopen("plot_script","w");

			fprintf(bash_script, "end_time=%d\n",j);

			//fprintf(bash_script, "set xrange[%lf:%lf]\nset yrange[%lf:%lf]\nunset key\n", 
			//	-0.5,MAX_CUBE_DIM+0.5,MIN_CUBE_DIM-0.5,MAX_CUBE_DIM+0.5);
			fprintf(bash_script, "set xrange[%lf:%lf]\nset yrange[%lf:%lf]\nunset key\n", 
				MIN_CUBE_DIM-0.5,MAX_CUBE_DIM+0.5,MIN_CUBE_DIM-0.5,MAX_CUBE_DIM+0.5);

			fprintf(bash_script, "set size square\n");
			fprintf(bash_script, "do for [i = 1:end_time]{\n  plot 'edge_file' using 1:2 w l lc 'black', \\\n");
			for(k = 0; k < (N-1); k++) fprintf(bash_script,"    'output.csv' using %d:%d:%d every ::i::i w circles lc %d fillstyle solid, \\\n",3*k+1,3*k+2,3*k+3,k);
			k=(N-1);
			//fprintf(bash_script,"    'output.csv' using %d:%d:%d every ::::i w lines lc %d \n",   3*k+1,3*k+2,3*k+3,k);
			fprintf(bash_script,"    'output.csv' using %d:%d:%d every ::i::i w circles lc %d fillstyle solid \n",   3*k+1,3*k+2,3*k+3,k);

			fprintf(bash_script,"   pause 0.1 \n } ");
			fclose(bash_script);
		}
	}
	else
	{
		if(DIMENSION > 2)
		{
			fprintf(fp, "c, %d, %d, %lf, %lf, %lf, %lf, %lf, %lf, %lf, %lf, %lf, %lf, %lf, %lf, %lf, %lf, %lf, %lf %lf\n", 
				i, j, time, 
				p_CPU[i].x, p_CPU[i].y, p_CPU[i].z, 
				0.0, 0.0, 0.0,
				v_CPU[i].x, v_CPU[i].y, v_CPU[i].z,
				collision_normal.x, collision_normal.y, collision_normal.z, 
				pressure.latest_value, gas_temperature.latest_value, heat_sum, entropy_sum
			);
		}
	}
}

void n_body()
{

	FILE * out_file;
	FILE * complex_event_log_file;
	char dir[256];

	int i, j, step, w;
	int smart_max_steps = MAX_STEPS;

	float3 v_in, v_out;

	bool complex_collisions_occurred = false;


	//WRITE INITIAL CONDITION TO FILE
	complex_event_log_file = fopen(strcat(strcpy(dir, dir_name), "complex_events_log.txt"), "w");
	out_file = fopen(strcat(strcpy(dir, dir_name), "output.csv"), "w");
	for(i = 0; i < N; i++) print_output_file(out_file, i, smart_max_steps, i, t_tot, collision_normal);

	step = 0;
	while(step++ <= smart_max_steps)
	{
	  	// on GPU - find smallest time step s.t. any particle(s) collide either 
	  	// with each other or a wall and update all particles to that time step
		find_dts<<<grid, block>>>(p_GPU, v_GPU, radius_GPU, mass_GPU, walls_GPU, tag_GPU, how_many_p_GPU, how_many_w_GPU, what_p_GPU, what_w_GPU, N, max_complex, ignore_particle_interaction, dt_GPU);
		errorCheck(step, "find_dts");

		//copy minimum time step and index of corresponding colliding element onto CPU 
		hipMemcpy( how_many_p_CPU, how_many_p_GPU,             N * sizeof(int  ), hipMemcpyDeviceToHost);
		hipMemcpy( how_many_w_CPU, how_many_w_GPU,             N * sizeof(int  ), hipMemcpyDeviceToHost);
		hipMemcpy(     what_p_CPU,     what_p_GPU,max_complex* N * sizeof(int  ), hipMemcpyDeviceToHost);
		hipMemcpy(     what_w_CPU,     what_w_GPU,max_complex* N * sizeof(int  ), hipMemcpyDeviceToHost);
		hipMemcpy(         dt_CPU,         dt_GPU,             N * sizeof(float), hipMemcpyDeviceToHost);


		// check (and modify tags) if complex collision events occurred
		complex_collisions_occurred = detect_collision_events();

		// if no collisions were detected, we are done. 
		if(dt_step < 0.0)
		{
			printf("\nEarly exit : dt_step = %f < 0 at step %i\n", dt_step, step);
			exit(1);
		}
		t_tot += dt_step;

		print_output_file(out_file,0,0,-4,t_tot,collision_normal);

		// update all particles to new time step
		for(i = 0; i < N; i++)
		{
			// update particle's position
			p_CPU[i] += v_CPU[i] * dt_step;

			// check if it is involved in a collision with either particle or wall and update velocity accordingly
			if( how_many_w_CPU[i] > 0)
			{
				v_in = v_CPU[i];
				for(j = 0; j < how_many_w_CPU[i]; j++)
				{
					w = -(1 + what_w_CPU[max_complex * i + j]);
					//end
					if(walls_CPU[w].type == end)
					{
						printf("step %d particle %d escaped through end wall\n", step, i);
						step = smart_max_steps + 2;
						i = N;
						j = num_walls;
						break;
					}
					v_out = walls_CPU[w].resolve_collision(i, v_in);
					tag_CPU[i * max_complex + j] = what_w_CPU[max_complex * i + j];
					v_in = v_out;
				}
				v_CPU[i] = v_out;
				print_output_file(out_file, i, w, step, t_tot, collision_normal);
			}
			else if( how_many_p_CPU[i] > 0)
			{
				j = what_p_CPU[max_complex * i];
				if(i > j) 
				{
					if(no_slip_particle) no_slip_particle_particle_collision(i,j);
					else elastic_particle_particle_collision(i, j);
				}
				print_output_file(out_file, i, j, step, t_tot, collision_normal);
			}
		}

		if( complex_collisions_occurred ) 
		{
			for(i = 0; i < complex_colliders; i++)
			{
				randomize_position(complex_event_particle[i], MIN_CUBE_DIM, MAX_CUBE_DIM);
				fprintf(complex_event_log_file, "%d, ", complex_event_log[i]);
			}
			fprintf(complex_event_log_file, "\n");
		}
		
		// update position on GPU to new time step
		// update velocity on GPU to match CPU 
		// (and also tag which keeps track of most recent collision for each particle)
		hipMemcpy(   p_GPU,   p_CPU,			N * sizeof(float3), hipMemcpyHostToDevice );
		hipMemcpy(   v_GPU,   v_CPU,			N * sizeof(float3), hipMemcpyHostToDevice );
		hipMemcpy( tag_GPU, tag_CPU, max_complex *	N * sizeof(int   ), hipMemcpyHostToDevice );

		//fireproofing: check at each time step that no particles escaped.
		check_no_particles_escape(step, t_tot);

		//end of this step
	}
	
	
	/*/  WRITE FINAL CONDITIONS TO FILE /*/
	for(i = 0; i < N; i++) print_output_file(out_file, i, 0, step, t_tot, collision_normal);

	fclose(out_file);
	fclose(complex_event_log_file);
	printf("%i gas particles, %d steps, %.4f seconds in time\n", N, step, t_tot);
}


int main(int argc, char** argv)
{
	clock_t time_0, time_1;
	FILE *fp;
	char dir[256];

	if(--argc < 1)
	{
		printf("Without input file, reverting to default parameters\n");
		no_input_file = true;
	}
	else
	{
		in_fname = argv[1];
		no_input_file = false;
	}

	time_0 = clock();

	read_input_file_and_set_initial_conditions();

    	n_body();
	time_1 = clock();

	printf("\n Runtime %.5f seconds\n", (float)(time_1 - time_0) / CLOCKS_PER_SEC);
	printf("\n DONE \n");

	fp = fopen(strcat(strcpy(dir, dir_name), "log"), "w");
	fprintf(fp, "N, Nsteps, physical_time, runtime, end_pressure, end_entropy_rate\n%d, %d, %lf, %lf, %lf, %lf", N, MAX_STEPS, t_tot, (float)(time_1 - time_0) / CLOCKS_PER_SEC, pressure.latest_value, entropy_sum / t_tot); 
	fclose(fp);

	return 0;
}
